#include "hip/hip_runtime.h"
#define PY_ARRAY_UNIQUE_SYMBOL cool_ARRAY_API
#define NO_IMPORT_ARRAY

#include <stdio.h>
#include <stdlib.h>
#include "Python.h"
#include "math.h"
#include "numpy/arrayobject.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/* Needs to be compiled as C files because of the Naming problem in Namespace */
#ifdef __cplusplus
extern "C" {
#endif

/*
    Calculate the new centers in the received block.
*/


#define THREAD_NUM   512
#define BLOCK_NUM   128
#define DATA_SIZE  THREAD_NUM*BLOCK_NUM

__global__ void chunk_centers_sum_cuda(double *cu_data,double *cu_centers, int* cu_centers_counter, double* cu_new_centers, int* cu_data_assigns, int* cluster_size,int *dimension)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int i,j,k;

    for(k = bid * THREAD_NUM + tid; k < DATA_SIZE; k+= BLOCK_NUM * THREAD_NUM)
    {
        double min_distance = 1E100;
        double distance;
        *(cu_data_assigns+k)=0;
        for (i = 0; i < *cluster_size; i++)
        {
            distance = 0;
            for (j = 0; j < *dimension; j++)
            {
                distance +=(*(cu_data+k*(*dimension)+j)-*(cu_centers+i*(*dimension)+j)) * (*(cu_data+k*(*dimension)+j)-*(cu_centers+i*(*dimension)+j));
            }
            if (distance <= min_distance)
            {
                min_distance = distance;
                *(cu_data_assigns+k) = i;
            }
        }
        __syncthreads();

        if(tid == 0)
        {
            *(cu_centers_counter+bid)=0;
            for (i=0 ; i<THREAD_NUM; i++)
            {
                *(cu_centers_counter+bid*(*cluster_size)+*(cu_data_assigns+bid*THREAD_NUM+i))+=1;
                for (j = 0; j < *dimension; j++)
                {
                    *(cu_new_centers +bid*(*cluster_size)*(*dimension) +(*(cu_data_assigns+bid*THREAD_NUM+i))* (*dimension) + j) += *(cu_data+(bid*THREAD_NUM+i)*(*dimension)+j);
                }
            }
           // printf("%d %d %d\n", *(cu_centers_counter+bid*(*cluster_size)),*(cu_centers_counter+bid*(*cluster_size)+1),*(cu_centers_counter+bid*(*cluster_size)+2));

        }
    }
}


PyObject* kmeans_chunk_center_cuda(PyArrayObject *data, PyArrayObject *centers, PyObject *data_assigns)
{
    /* Record the nearest center of each point and renew the centers with the points near one given center. */
    int cluster_size, dimension, chunk_size;
    cluster_size = *(int *)PyArray_DIMS(centers);
    dimension = PyArray_DIM(centers, 1);
    chunk_size = *(int *)PyArray_DIMS(data);
    int *centers_counter = (int *)malloc(sizeof(int) * BLOCK_NUM* cluster_size);
    double *new_centers = (double *)malloc(sizeof(double)* BLOCK_NUM * cluster_size * dimension);
    int* p_data_assigns= (int *)malloc(sizeof(int) * chunk_size);

    int i, j, k;

    for (i = 0; i < cluster_size* BLOCK_NUM; i++)
    {
	    (*(centers_counter + i)) = 0;
    }

    for (i = 0; i < cluster_size * dimension* BLOCK_NUM; i++)
    {
	    (*(new_centers + i)) = 0;
    }

    double* p_data=(double *)PyArray_DATA(data);
    double* p_centers=(double *)PyArray_DATA(centers);

    double* cu_data, *cu_centers, *cu_new_centers;
    int* cu_centers_counter, *cu_cluster_size, *cu_dimension, *cu_data_assigns;

    hipMalloc((void**) &cu_data, sizeof(double) * chunk_size * dimension);
    hipMalloc((void**) &cu_centers, sizeof(double) * cluster_size * dimension);
    hipMalloc((void**) &cu_centers_counter, sizeof(int) * BLOCK_NUM * cluster_size);
    hipMalloc((void**) &cu_new_centers, sizeof(double) * BLOCK_NUM * cluster_size * dimension);
    hipMalloc((void**) &cu_data_assigns, sizeof(int) * chunk_size );
    hipMalloc((void**) &cu_cluster_size, sizeof(int) *1);
    hipMalloc((void**) &cu_dimension, sizeof(int) *1);

    hipMemcpy(cu_data, p_data, sizeof(double) * chunk_size * dimension, hipMemcpyHostToDevice);
    hipMemcpy(cu_centers, p_centers, sizeof(double) * cluster_size * dimension, hipMemcpyHostToDevice);
    hipMemcpy(cu_centers_counter, centers_counter, sizeof(int)* BLOCK_NUM * cluster_size,hipMemcpyHostToDevice);
    hipMemcpy(cu_new_centers, new_centers, sizeof(double) * BLOCK_NUM * cluster_size * dimension, hipMemcpyHostToDevice);
    hipMemcpy(cu_data_assigns, p_data_assigns, sizeof(int) *chunk_size , hipMemcpyHostToDevice);
    hipMemcpy(cu_cluster_size, &cluster_size, sizeof(int) * 1, hipMemcpyHostToDevice);
    hipMemcpy(cu_dimension, &dimension, sizeof(int) * 1, hipMemcpyHostToDevice);


    chunk_centers_sum_cuda<<<BLOCK_NUM, THREAD_NUM>>>(cu_data,cu_centers,cu_centers_counter,cu_new_centers,cu_data_assigns,cu_cluster_size,cu_dimension);

    hipMemcpy(centers_counter, cu_centers_counter,sizeof(int) * BLOCK_NUM *cluster_size, hipMemcpyDeviceToHost);
    hipMemcpy(new_centers, cu_new_centers, sizeof(double) * BLOCK_NUM* cluster_size * dimension, hipMemcpyDeviceToHost);
    hipMemcpy(p_data_assigns, cu_data_assigns, sizeof(int) * chunk_size  , hipMemcpyDeviceToHost);

    for (i=0; i<BLOCK_NUM ;i++)
    {
        for (j=0;j<cluster_size; j++)
        {
            *(centers_counter+j)+=*(centers_counter+i*cluster_size+j);
        }
    }
    for (i=0; i<BLOCK_NUM ;i++)
    {
        for (j=0;j<cluster_size; j++)
        {
            for (k=0;k<dimension;k++)
            *(new_centers+j*dimension+k)+=*(new_centers+i*cluster_size*dimension+j*dimension+k);
        }
    }

    hipFree(cu_data);
    hipFree(cu_centers);
    hipFree(cu_centers_counter);
    hipFree(cu_new_centers);
    hipFree(cu_data_assigns);
    hipFree(cu_cluster_size);
    hipFree(cu_dimension);


    for (i = 0; i < cluster_size; i++)
    {
        if (*(centers_counter + i) == 0)
        {
            for (j = 0; j < dimension; j++)
            {
                (*(new_centers + i * dimension + j)) = (*(double*)PyArray_GETPTR2(centers, i, j));
            }
        }
        else
        {
            for (j=0; j < dimension; j++)
            {
                (*(new_centers + i * dimension + j)) /= (*(centers_counter + i));
                //printf("%lf ",(*(new_centers + i * dimension + j)) );
            }
        }
    }

    for (i=0; i<chunk_size; i++)
    {
        PyList_SetItem(data_assigns, i, PyInt_FromLong(*(p_data_assigns+i)));
    }

    PyObject* return_new_centers;
    npy_intp dims[2] = {cluster_size, dimension};
    return_new_centers = PyArray_SimpleNew(2, dims, NPY_DOUBLE);
    void *arr_data = PyArray_DATA((PyArrayObject*)return_new_centers);
    memcpy(arr_data, new_centers, PyArray_ITEMSIZE((PyArrayObject*) return_new_centers) * cluster_size * dimension);
    /* Need to copy the data of the malloced buffer to the PyObject
       since the malloced buffer will disappear after the C extension is called. */
    free(centers_counter);
    free(new_centers);
    free(p_data_assigns);
    return (PyObject*) return_new_centers;
}

#ifdef __cplusplus
}
#endif
